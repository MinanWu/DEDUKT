#include "hip/hip_runtime.h"
#include "UniversalHittingSet_Hashtable.hpp"

#include <fstream>



uhs_hashtable_slot* new_uhs_hashtable(int hashtable_capacity, int myrank) {
	uhs_hashtable_slot* hashtable;
	size_t hashtable_size = (hashtable_capacity * sizeof(uhs_hashtable_slot));
	hipMalloc(&hashtable, hashtable_size);
	hipMemset(&hashtable, 0, hashtable_size);
	return hashtable;
}



std::string uhs_string_from_file(char* uhs_file_path) {

	std::string uhs_string;

	std::ifstream uhs_file(uhs_file_path);
	if (!(uhs_file.is_open())) {
		printf("Error: Failed to open UHS file '%s'\n", uhs_file_path);
		return uhs_string;
	}

	// std::string uhs_string(std::istreambuf_iterator<char>{uhs_file}, {});
	std::string uhs_file_line;
	while (!(uhs_file.eof())) {
		getline(uhs_file, uhs_file_line);
		uhs_string.append(uhs_file_line);
	}
	uhs_file.close();

	return uhs_string;

}



__device__ uhs_key_t mmer_numeric_at_gpu(int start_index, int m, const char* uhs_string) {
	uhs_key_t mmer_numeric = 0;
	for (int offset = 0; offset < m; offset += 1) {
		char s = uhs_string[start_index + offset];
		size_t x = ((s & 4) >> 1);
		mmer_numeric |= ((x + ((x ^ (s & 2)) >> 1)) << ((2 * (m - 1)) - (2 * offset)));
	}
	return mmer_numeric;
}


__global__ void populate_uhs_hashtable_keys_gpu(const char* uhs_string, int MINIMIZER_LENGTH, int mmers_count, uhs_hashtable_slot* uhs_hashtable, uint64_t uhs_hashtable_capacity, uhs_key_t* uhs_mmers, uint64_t uhs_mmers_count) {

	unsigned int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thread_id >= mmers_count) {
		return;
	}

	int mmer_start_index = (thread_id * MINIMIZER_LENGTH);
	uhs_key_t mmer_numeric = mmer_numeric_at_gpu(mmer_start_index, MINIMIZER_LENGTH, uhs_string);

	uhs_mmers[thread_id] = mmer_numeric;

	uint32_t slot_index = (cuda_murmur3_64(mmer_numeric) & (uhs_hashtable_capacity - 1));

	while (true) {
		uhs_key_t old_key = atomicCAS(&(uhs_hashtable[slot_index].key), uhs_key_empty, mmer_numeric);
		if (old_key == uhs_key_empty || old_key == mmer_numeric) {
			return;
		}
		slot_index = ((slot_index + 1) & (uhs_hashtable_capacity - 1));
	}

}


uhs_hashtable_slot* initialize_uhs_frequencies_hashtable(char* uhs_file_path, int MINIMIZER_LENGTH, int myrank, uint64_t* output_uhs_hashtable_capacity, uhs_key_t* output_uhs_mmers, uint64_t* output_uhs_mmers_count) {

	std::string uhs_string = uhs_string_from_file(uhs_file_path);

	char* uhs_string_gpu;
	size_t uhs_string_size = ((uhs_string.length() + 1) * sizeof(char));
	hipMalloc(((void**) &uhs_string_gpu), uhs_string_size);
	hipMemcpy(uhs_string_gpu, ((char*) uhs_string.data()), uhs_string_size, hipMemcpyHostToDevice);

	int uhs_mmers_count = (uhs_string.length() / MINIMIZER_LENGTH);

	uhs_key_t* uhs_mmers;
	hipMalloc(((void**) &uhs_mmers), (uhs_mmers_count * sizeof(uhs_key_t)));

	uint64_t uhs_hashtable_capacity = 1;
	while (uhs_hashtable_capacity < ((uint64_t) mmers_count)) {
		uhs_hashtable_capacity *= 2;
	}

	uhs_hashtable_slot* uhs_frequencies_hashtable = new_uhs_hashtable(uhs_hashtable_capacity, myrank);

	int min_grid_size;
	int thread_block_size;
	hipOccupancyMaxPotentialBlockSize(
		&min_grid_size,
		&thread_block_size,
		populate_uhs_hashtable_keys_gpu,
		0,
		0
	);
	int grid_size = ((mmers_count + (thread_block_size - 1)) / thread_block_size);

	populate_uhs_hashtable_keys_gpu<<<grid_size, thread_block_size>>>(
		uhs_string_gpu,
		MINIMIZER_LENGTH,
		mmers_count,
		uhs_frequencies_hashtable,
		uhs_hashtable_capacity,
		uhs_mmers,
		uhs_mmers_count
	);
	hipDeviceSynchronize();

	hipFree(uhs_string_gpu);

	*output_uhs_hashtable_capacity = uhs_hashtable_capacity;
	*output_uhs_mmers = uhs_mmers;
	*output_uhs_mmers_count = uhs_mmers_count;
	return uhs_frequencies_hashtable;

}



__device__ uhs_value_t get_mmer_frequency_gpu(uhs_key_t mmer_numeric, uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity) {
	uint32_t slot_index = (cuda_murmur3_64(mmer_numeric) & (uhs_hashtable_capacity - 1));
	while (true) {
		uhs_key_t slot_key = uhs_frequencies_hashtable[slot_index].key;
		if (slot_key == mmer_numeric) {
			uhs_value_t frequency = uhs_frequencies_hashtable[slot_index].value;
			return frequency;
		}
		else if (slot_key == uhs_key_empty) {
			return uhs_value_empty;
		}
		slot_index = ((slot_index + 1) & (uhs_hashtable_capacity - 1));
	}
	return uhs_value_empty;
}



__device__ void increment_mmer_frequency_gpu(uhs_key_t mmer_numeric, uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity) {
	uint32_t slot_index = (cuda_murmur3_64(mmer_numeric) & (uhs_hashtable_capacity - 1));
	while (true) {
		uhs_key_t slot_key = uhs_frequencies_hashtable[slot_index].key;
		if (slot_key == mmer_numeric) {
			atomicAdd(&(uhs_frequencies_hashtable[slot_index].value), 1);
			return;
		}
		else if (slot_key == uhs_key_empty) {
			return;
		}
		slot_index = ((slot_index + 1) & (uhs_hashtable_capacity - 1));
	}
}


__global__ void set_uhs_frequencies_from_sample_gpu(char* sequence, unsigned int sequence_length, double sample_percentage, int MINIMIZER_LENGTH, uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity) {
	unsigned int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int sample_gap = std::ceil(1.0 / sample_percentage);
	int mmer_start_index = (thread_id * sample_gap);
	if ((mmer_start_index + MINIMIZER_LENGTH) > sequence_length) {
		return;
	}
	uhs_key_t mmer_numeric = mmer_numeric_at_gpu(mmer_start_index, MINIMIZER_LENGTH, ((const char*) sequence));
	increment_mmer_frequency_gpu(mmer_numeric, uhs_frequencies_hashtable, uhs_hashtable_capacity);
}


void set_uhs_frequencies_from_sample(char* sequence, unsigned int sequence_length, double sample_fraction, int MINIMIZER_LENGTH, uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity) {

	int min_grid_size;
	int thread_block_size;
	hipOccupancyMaxPotentialBlockSize(
		&min_grid_size,
		&thread_block_size,
		set_uhs_frequencies_from_sample_gpu,
		0,
		0
	);
	int grid_size = ((std::ceil(sequence_length * sample_fraction) + (thread_block_size - 1)) / thread_block_size);

	set_uhs_frequencies_from_sample_gpu<<<grid_size, thread_block_size>>>(
		sequence,
		sequence_length,
		sample_fraction,
		MINIMIZER_LENGTH,
		uhs_frequencies_hashtable,
		uhs_hashtable_capacity
	);
	hipDeviceSynchronize();

}



__global__ void reset_uhs_hashtable_frequencies_gpu(uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity, uhs_key_t uhs_mmers, uint64_t uhs_mmers_count) {
	
	unsigned int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thread_id >= uhs_mmers_count) {
		return;
	}

	uhs_key_t mmer_numeric = uhs_mmers[thread_id];

	uint32_t slot_index = (cuda_murmur3_64(mmer_numeric) & (uhs_hashtable_capacity - 1));
	while (true) {
		keyType slot_key = uhs_frequencies_hashtable[slot_index].key;
		if (slot_key == mmer_numeric) {
			uhs_frequencies_hashtable[slot_index].value = 0;
			return;
		}
		else if (slot_key == uhs_key_empty) {
			return;
		}
		slot_index = ((slot_index + 1) & (uhs_hashtable_capacity - 1));
	}

}


void reset_uhs_hashtable_frequencies(uhs_hashtable_slot* uhs_frequencies_hashtable, uint64_t uhs_hashtable_capacity, uhs_key_t uhs_mmers, uint64_t uhs_mmers_count) {
	
	int min_grid_size;
	int thread_block_size;
	hipOccupancyMaxPotentialBlockSize(
		&min_grid_size,
		&thread_block_size,
		set_uhs_frequencies_from_sample_gpu,
		0,
		0
	);
	int grid_size = ((uhs_mmers_count + (thread_block_size - 1)) / thread_block_size);

	reset_uhs_hashtable_frequencies_gpu<<<grid_size, thread_block_size>>>(
		uhs_frequencies_hashtable,
		uhs_hashtable_capacity,
		uhs_mmers,
		uhs_mmers_count
	);

}