#include "hip/hip_runtime.h"
#include "Kmer.hpp"
#include "Friends.h"
#include "MPIType.h"
#include "SimpleCount.h"
#include "FriendsMPI.h"
#include "Pack.h"
#include "supermer.h"


// int MINIMIZER_LENGTH = 5;
// int mlen = MINIMIZER_LENGTH;

__device__ keyType find_minimizer(keyType kmer, int &order, int klen, int mlen, keyType max64){
	
	keyType minimizer = max64;
	int local_order;

	for (int m = 0; m < (klen - mlen + 1); ++m){
		keyType mmer = ((kmer >> (2*(31-(m+mlen-1)))) );
		
		if( mmer < minimizer ) {
			local_order = m;
			minimizer = mmer;
		}
	}
	order += local_order;
	return minimizer;
}

__device__ keyType find_minimizer(char* kmer, int &order, int klen, int mlen){
	
	char *minimizer = "ZZZZZ";
	keyType com_mini = 0;/////////INT_MAX;
	// int local_order;
	// char mmer[5];
	
	// for (int m = 0; m < (klen - mlen); ++m){
		
	// 	for (int i = 0; i < mlen; ++i)
	// 		mmer[i] = kmer[m+i];
	// 	if( mmer < minimizer ) {
	// 		local_order = m;
	// 		minimizer = mmer;
	// 	}
	// }
	// order += local_order;

	// for (int m = 0; m < mlen; ++m) {
 //        char s = mmer[m];//minimizer[m];
 //        int j = m % 32;
 //        size_t x = ((s) & 4) >> 1;
 //        com_mini |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j))); //make it longs[] to support larger kmer 	
 //    }

	return com_mini;
}

// size_t total_kmers = 0, total_supermers = 0, tot_char = 0;



__global__ void cuda_build_supermer(char *seq, char *kmers, int klen, int mlen, unsigned int seq_len,
    keyType* outgoing, unsigned char *out_slen, int *owner_counter, int nproc, unsigned int p_buff_len, int per_block_seq_len){
    
    unsigned int tId = threadIdx.x;
    unsigned int laneId = tId & (blockDim.x - 1);
    unsigned int gId = (blockIdx.x * blockDim.x + tId);
    int window = klen - mlen + 1 ;    
    // int per_block_seq_len = blockDim.x * window;   
    // int per_block_seq_len = (seq_len + (gridDim.x - 1)) / gridDim.x;
    int st_char_block = blockIdx.x * per_block_seq_len; //first char this block should read
    int nKmer = seq_len - klen + 1; //last char is 'a'
    keyType max64 = 18446744073709551615;

    for(int i = st_char_block + laneId * window; i < (st_char_block + per_block_seq_len) && i < nKmer ; i+=blockDim.x*window) {
     // if(tId == 0)
     //    printf("GPU smer info %d %d %d %d \n", gridDim.x,  blockIdx.x, per_block_seq_len, st_char_block );  
 
        keyType longs = 0; //GPU CAS support this for 64 bit
        bool validKmer = true, inserted = false;
        int slen = klen;
        keyType comprs_Smer = 0;
        keyType comprs_Kmer = 0;

        keyType cur_mini = max64;  keyType prev_mini = cur_mini;	
        
        int order = 0, prev_order = 0;
        
        for (int w = 0; w < window && (i+ w + klen) < nKmer; ++w){ //make it linear time	
       		inserted = false; validKmer = true;
        	
        	for (int k = 0; k < klen ; ++k) {
	            char s =  seq[i + w + k ];
	            if(s == 'a' || s == 'N')  { //improvement scope..discard a chunk based on loc of N
	                validKmer = false; break;
	            }
	            int j = k % 32;
	            size_t x = ((s) & 4) >> 1;
	            comprs_Kmer |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j))); //make it longs[] to support larger kmer 	
	            order = i + w;
	            // cur_kmer[k] = seq[i + w + k ];
	        }
			if(validKmer)
            {
                if(w == 0) //
                { //not initilized yet with any mini
                    // cur_mini = find_minimizer(cur_kmer, order, klen, mlen);//find_minimizer(comprs_Kmer, order, klen, mlen);
                    cur_mini = find_minimizer(comprs_Kmer, order, klen, mlen, max64);
                    comprs_Smer = comprs_Kmer; slen = klen;
                     // if(gId == 0) printf(" brand new smer %lu %lu \n", cur_mini, comprs_Smer );   
                }
                else 
                {
		  	     	
                    cur_mini = find_minimizer(comprs_Kmer, order, klen, mlen, max64);;//find_minimizer(cur_kmer, order, klen, mlen);
		        	
		        	if(prev_mini == cur_mini ){	
                        // printf("mini match  %lu %lu \n", cur_mini, comprs_Smer ); 	     
			        	char s =  seq[i + w + klen - 1];
			            int j = slen % 32; 
			            size_t x = ((s) & 4) >> 1;
			            comprs_Smer |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j)));
			            slen++;
		        	}
			     	else 
                    {		                  
		            	keyType owner = cuda_murmur3_64(comprs_Smer) & (nproc - 1); // remove & with HTcapacity in func
		            	int old_count = atomicAdd(&owner_counter[owner],1); 
			            // if(gId == 0) printf("%lu %d\n", owner, old_count );
                        if(old_count >= p_buff_len )  { 
                            printf("%d %d \n", old_count, p_buff_len );
                            printf("Overflow!! MISSION ABORT!!\n");
                        }           	
                        outgoing[owner * p_buff_len + old_count] = comprs_Smer; //hash (longs) 
		            	out_slen[owner * p_buff_len + old_count] = slen;  
		            	comprs_Smer = comprs_Kmer;
                        slen = klen;
		            	// if(w == (window - 1) || (i == nKmer - 1)) inserted = true;
			        }
			    }
			    prev_mini = cur_mini;
		    	prev_order = order;
			}
			// else cur_mini == max64;
	    }
	    if(validKmer)
        {
	    	keyType owner = cuda_murmur3_64(comprs_Smer) & (nproc - 1); // remove & with HTcapacity in func
        	int old_count = atomicAdd(&owner_counter[owner],1); 
            if(old_count >= p_buff_len)   printf("Overflow!! MISSION ABORT!!\n");     
        	outgoing[owner * p_buff_len + old_count] = comprs_Smer; //hash (longs)
        	out_slen[owner * p_buff_len + old_count] = slen;   
	    }
    }
}
int buff_scale = 2;

void getSupermers_GPU(char* seq, int klen, int mlen, int nproc, int *owner_counter, 
	keyType* h_send_smers, unsigned char* h_send_slens, int n_kmers, int rank )
{
    klen= 17;
	int count, devId;
    char *d_kmers, *d_seq;
    keyType *d_supermers, *d_outOverflowBuff;
    unsigned char *d_slen;
    int *d_owner_counter; 
        
    // Map MPI ranks to GPUs
    hipGetDeviceCount(&count);
    int gpuID = rank % count;
    hipSetDevice(gpuID);
    hipGetDevice(&devId);

    unsigned int seq_len = strlen(seq);
    if(seq_len < klen) return;// h_outgoing;
    // unsigned int n_kmers =  seq_len - klen + 1;

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    
    cuda_timer_start(start);


    // CUDA mallocs
    checkCuda (hipMalloc(&d_supermers, n_kmers * buff_scale * sizeof(keyType)), __LINE__);  // giving 2x space to each node 
    checkCuda (hipMalloc(&d_slen, n_kmers * buff_scale * sizeof(unsigned char)), __LINE__);  // giving 2x space to each node 
    checkCuda (hipMalloc(&d_seq, seq_len * sizeof(char)), __LINE__);
    checkCuda (hipMalloc(&d_owner_counter, nproc * sizeof(int)), __LINE__);
    // CUDA memcopies
    checkCuda (hipMemcpy(d_seq, seq, seq_len * sizeof(char) , hipMemcpyHostToDevice), __LINE__);
    hipMemset(d_supermers,  0, n_kmers * buff_scale * sizeof(keyType));
    hipMemset(d_owner_counter,  0, sizeof(int) * nproc);

	int window = klen - mlen + 1 ;
	unsigned int p_buff_len = ((n_kmers * buff_scale) + nproc - 1)/nproc;
    int b = 128;
    int g = (seq_len + (b -1) ) / (b);// * window;
    // g = (g+window-1)/window;
    int per_block_seq_len = b * window;// ((seq_len+window-1/window) + (g - 1)) / g;
    // cout << seq_len << "  info " << g << " " << per_block_seq_len << endl;

    // Kernel call
    cuda_build_supermer<<<g, b>>>(d_seq, d_kmers, klen, mlen, seq_len, d_supermers, d_slen, d_owner_counter, nproc, p_buff_len, per_block_seq_len);

    // h_outgoing = (keyType *) malloc ( n_kmers * buff_scale * sizeof(keyType));
    //***** copy back to CPU *****
    checkCuda (hipMemcpy(h_send_smers, d_supermers, n_kmers * buff_scale * sizeof(keyType), hipMemcpyDeviceToHost), __LINE__); 
    checkCuda (hipMemcpy(h_send_slens, d_slen, n_kmers * buff_scale * sizeof(unsigned char), hipMemcpyDeviceToHost), __LINE__); 
    checkCuda (hipMemcpy(owner_counter, d_owner_counter, nproc * sizeof(int) , hipMemcpyDeviceToHost), __LINE__); 
   
    size_t total_counter = 0;
    for (int i = 0; i < nproc; ++i) {   
        total_counter += owner_counter[i];    
        // printf("GPU Supermer pack: output buffer: %d %d \n", owner_counter[i], total_counter);
    }
    hipFree(d_seq);
    hipFree(d_supermers);
    hipFree(d_slen);
    hipFree(d_owner_counter);

    cuda_timer_stop(start, stop, milliseconds);
    return;
}


__global__ void cu_kcounter_smer(KeyValue* hashtable, const keyType* kvs, const unsigned char* slens,  unsigned int numkvs, int klen, keyType mask)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
   
    if (threadid < numkvs){

        keyType new_smer = kvs[threadid];
        unsigned char c = slens[threadid];
        int slen = (int)c;

           
        for(int k = 0; k < (slen - klen + 1); ++k)
        {
            keyType new_key = ((new_smer) >> (2*(31-(klen+k -1)))) & mask;//kvs[threadid];//.key;
             
            keyType slot = cuda_murmur3_64(new_key) & (kHashTableCapacity-1);
            
            while (true){
                keyType old_key = atomicCAS(&hashtable[slot].key, kEmpty, new_key);
                      
                if (old_key == kEmpty || old_key == new_key) {
                    atomicAdd(&hashtable[slot].value,1);
                    break;
                }
                slot = (slot + 1) & (kHashTableCapacity-1);
            }
        }
    }
}

void kcounter_supermer_GPU(KeyValue* pHashTable, keyType* d_smers, unsigned char* d_slen, uint32_t num_keys, int klen, int rank)
{
    // Map MPI ranks to GPUs
    int count, devId;
    hipGetDeviceCount(&count);
    int gpuID = rank % count;
    hipSetDevice(gpuID);
    hipGetDevice(&devId);
    // printf("\n FROnProcs %d: rank %d mapped to %d\n", nproc, rank, devId);

    // Copy the keyvalues to the GPU
    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	const uint32_t N = 1;
    /*------------------------
     Copy kmers to GPU      
    ------------------------*/
    // keyType* device_keys;
    hipEventRecord(start);

    int b = 128;
    int g= (N + (b - 1)) / b;
   
    /*----------------------------
    CUDA call: Insert kmers to HT       
    ------------------------------*/

    // Have CUDA calculate the thread block size
    keyType mask = pow(2, 2 * klen) - 1;
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, cu_kcounter_smer, 0, 0);


    unsigned char * h_slens = (unsigned char *) malloc(num_keys * sizeof(unsigned char));
    checkCuda (hipMemcpy(h_slens, d_slen, num_keys * sizeof(unsigned char), hipMemcpyDeviceToHost), __LINE__); 
   // keyType * h_smers = (keyType *) malloc(num_keys * sizeof(keyType));
   //  checkCuda (hipMemcpy(h_smers, d_smers, num_keys * sizeof(keyType), hipMemcpyDeviceToHost), __LINE__); 
   
   cout << "after memcopies: ";
for (int i = 0; i < num_keys; ++i)
{
    if(i < 10 ) cout << h_slens[i] << " ";
    // if(h_slens[i] > 17)
    // cout << i << ": " << h_slens[i] << " larger ";
}


    int gridsize = ((uint32_t)num_keys + threadblocksize - 1) / threadblocksize;
    cu_kcounter_smer<<<gridsize, threadblocksize>>>(pHashTable, d_smers, d_slen, (uint32_t)num_keys, klen, mask);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipFree(d_smers);
    hipFree(d_slen);
    return ;//h_pHashTable;

}


	