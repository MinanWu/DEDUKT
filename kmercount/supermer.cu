#include "hip/hip_runtime.h"
#include "Kmer.hpp"
#include "Friends.h"
#include "MPIType.h"
#include "SimpleCount.h"
#include "FriendsMPI.h"
#include "Pack.h"
#include "supermer.h"

__device__ keyType find_minimizer(keyType kmer, int klen, int mlen, keyType max64){

	keyType minimizer = max64;
	keyType mask = pow(2, 2 * mlen) - 1;

	for (int m = 0; m < (klen - mlen ); ++m){
		keyType mmer =  (kmer >> (2*(31-(mlen+m -1)))) & mask;

		if( mmer < minimizer ) 
			minimizer = mmer;
	}
	return minimizer;
}

__global__ void cuda_build_supermer(char *seq, char *kmers, int klen, int mlen, unsigned int seq_len,
		keyType* outgoing, unsigned char *out_slen, int *owner_counter, int nproc, unsigned int p_buff_len, 
		int per_block_seq_len, int window, int rank){

	unsigned int tId = threadIdx.x;
	unsigned int laneId = tId & (blockDim.x - 1);
	unsigned int gId = (blockIdx.x * blockDim.x + tId);
	int st_char_block = blockIdx.x * per_block_seq_len; //first char this block should read
	int nKmer = seq_len - klen + 1; //last char is 'a'
	keyType max64 = 18446744073709551615;

	bool validKmer = true;
	int slen = klen;
	keyType comprs_Smer = 0;
	keyType comprs_Kmer = 0;
	int owner = -1;
	int old_count=-1;

	keyType cur_mini = max64;  keyType prev_mini = cur_mini;    

	//****First kmer of this window *****
	int i = st_char_block + laneId * window;

	if(i <  nKmer) {

		comprs_Kmer = 0;
		for (int k = 0; k < klen ; ++k) {
			char s =  seq[i + k ];
			if(s == 'a' || s == 'N')  { 
				// w += klen-1;
				validKmer = false; break; //FIXIT can have 'n'
			}
			int j = k % 32;
			size_t x = ((s) & 4) >> 1;
			// comprs_Kmer |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j))); //make it longs[] to support larger kmer  

			switch(s) { //redefined
				case 'A': comprs_Kmer |= ((x + (x^1)) << (2*(31-j)));  break;
				case 'C': comprs_Kmer |= ((x + (x^0)) << (2*(31-j)));  break;
				case 'G': comprs_Kmer |= ((x + (x^3)) << (2*(31-j)));  break;
				case 'T': comprs_Kmer |= ((x + (x^2)) << (2*(31-j)));  break;
			}
		}
		if(validKmer){
			cur_mini = find_minimizer(comprs_Kmer, klen, mlen, max64);
			prev_mini = cur_mini; 
			// owner = cuda_murmur3_64(cur_mini) & (nproc - 1); // remove & with HTcapacity in func
			// keyType myhash = cuda_murmur3_64(cur_mini); // remove & with HTcapacity in func
			keyType myhash = cuda_MurmurHash3_x64_128((const void *)&cur_mini, 8, 313);// & (nproc - 1);
			double range = static_cast<double>(myhash) * static_cast<double>(nproc);
			owner = range / max64;

			old_count = atomicAdd(&owner_counter[owner],1); 
			outgoing[owner * p_buff_len + old_count] = comprs_Kmer; //hash (longs)
			out_slen[owner * p_buff_len + old_count] = klen;  

		}
		comprs_Smer = comprs_Kmer;
		slen = klen;

		int c = st_char_block + (laneId * window);

		for(int w = 1; w < window && (c+w) < nKmer ; w++) {

			validKmer = true;
			comprs_Kmer = 0;
			// if ((i + klen-1) > nKmer) return;
			for (int k = 0; k < klen ; ++k) {
				char s =  seq[c + w + k ];
				if(s == 'a' || s == 'N')  { 
					// w += klen-1;
					validKmer = false; break;
				}
				int j = k % 32;
				size_t x = ((s) & 4) >> 1;
				// comprs_Kmer |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j))); //make it longs[] to support larger kmer  
				switch(s) { //redefined
					case 'A': comprs_Kmer |= ((x + (x^1)) << (2*(31-j)));  break;
					case 'C': comprs_Kmer |= ((x + (x^0)) << (2*(31-j)));  break;
					case 'G': comprs_Kmer |= ((x + (x^3)) << (2*(31-j)));  break;
					case 'T': comprs_Kmer |= ((x + (x^2)) << (2*(31-j)));  break;
				}
			}  

			if(validKmer){ 

				cur_mini = find_minimizer(comprs_Kmer, klen, mlen, max64);

				if(prev_mini == cur_mini){ 
					// printf("mini match  %lu %lu \n", cur_mini, comprs_Smer );         
					char s =  seq[c + w + klen - 1];
					int j = slen % 32; 
					size_t x = ((s) & 4) >> 1;
					// comprs_Smer |= ((x + ((x ^ (s & 2)) >>1)) << (2*(31-j)));
					switch(s) { //redefined
						case 'A': comprs_Smer |= ((x + (x^1)) << (2*(31-j)));  break;
						case 'C': comprs_Smer |= ((x + (x^0)) << (2*(31-j)));  break;
						case 'G': comprs_Smer |= ((x + (x^3)) << (2*(31-j)));  break;
						case 'T': comprs_Smer |= ((x + (x^2)) << (2*(31-j)));  break;
					}
					slen++;
				}
				else 	{ 

					if(owner > -1 && old_count > -1)
					{
						outgoing[owner * p_buff_len + old_count] = comprs_Smer; //hash (longs) 
						out_slen[owner * p_buff_len + old_count] = slen;                          
					}
					//* new supermer */
					slen = klen;
					comprs_Smer = comprs_Kmer;
					// owner = cuda_murmur3_64(cur_mini) & (nproc - 1); // remove & with HTcapacity in func
					keyType myhash = cuda_MurmurHash3_x64_128((const void *)&cur_mini, 8, 313);
					// keyType myhash = cuda_murmur3_64(cur_mini); // remove & with HTcapacity in func
					double range = static_cast<double>(myhash) * static_cast<double>(nproc);
					owner = range / max64;

					old_count = atomicAdd(&owner_counter[owner],1); 
					if(old_count > p_buff_len )  { 
						printf("Overflow!! MISSION ABORT!!\n"); return;
					}               
					outgoing[owner * p_buff_len + old_count] = comprs_Smer; //hash (longs) 
					out_slen[owner * p_buff_len + old_count] = slen;  
				}
				prev_mini = cur_mini;
			}
		}   
		if(old_count > -1 && owner > -1) {
			outgoing[owner * p_buff_len + old_count] = comprs_Smer; //hash (longs)
			out_slen[owner * p_buff_len + old_count] = slen; 
		}
	}         
}

void getSupermers_GPU(char* seq, int klen, int mlen, int nproc, int *owner_counter, 
		keyType* h_send_smers, unsigned char* h_send_slens, int n_kmers, int rank, int BUFF_SCALE )
{

	int count, devId;
	char *d_kmers, *d_seq;
	keyType *d_supermers, *d_outOverflowBuff;
	unsigned char *d_slen;
	int *d_owner_counter; 

	//* Map MPI ranks to GPUs */
	hipGetDeviceCount(&count);
	int gpuID = rank % count;
	hipSetDevice(gpuID);
	hipGetDevice(&devId);

	unsigned int seq_len = strlen(seq);
	if(seq_len < klen) return;// h_outgoing;

	//* Create events for GPU timing */
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	cuda_timer_start(start);

	// CUDA mallocs
	checkCuda (hipMalloc(&d_supermers, n_kmers * BUFF_SCALE * sizeof(keyType)), __LINE__);  // giving 2x space to each node 
	checkCuda (hipMalloc(&d_slen, n_kmers * BUFF_SCALE * sizeof(unsigned char)), __LINE__);  // giving 2x space to each node 
	checkCuda (hipMalloc(&d_seq, seq_len * sizeof(char)), __LINE__);
	checkCuda (hipMalloc(&d_owner_counter, nproc * sizeof(int)), __LINE__);
	// CUDA memcopies
	checkCuda (hipMemcpy(d_seq, seq, seq_len * sizeof(char) , hipMemcpyHostToDevice), __LINE__);
	hipMemset(d_supermers,  0, n_kmers * BUFF_SCALE * sizeof(keyType));
	hipMemset(d_owner_counter,  0, sizeof(int) * nproc);

	int window = 32 - klen;// - mlen + 1 ;

	unsigned int p_buff_len = ((n_kmers * BUFF_SCALE) + nproc - 1)/nproc;

	int b = 128;
	int g = (seq_len + (b*window - 1) ) / (b*window); ;//(seq_len + (b -1) ) / (b);// * window;
	int per_block_seq_len = b * window;// ((seq_len+window-1/window) + (g - 1)) / g;

	// Kernel call
	cuda_build_supermer<<<g, b>>>(d_seq, d_kmers, klen, mlen, seq_len, d_supermers, d_slen, d_owner_counter, nproc, p_buff_len, per_block_seq_len, window, rank);

	//* Memcopy to CPU */
	checkCuda (hipMemcpy(h_send_smers, d_supermers, n_kmers * BUFF_SCALE * sizeof(keyType), hipMemcpyDeviceToHost), __LINE__); 
	checkCuda (hipMemcpy(h_send_slens, d_slen, n_kmers * BUFF_SCALE * sizeof(unsigned char), hipMemcpyDeviceToHost), __LINE__); 
	checkCuda (hipMemcpy(owner_counter, d_owner_counter, nproc * sizeof(int) , hipMemcpyDeviceToHost), __LINE__); 

	// size_t total_counter = 0;
	// cout << rank << " smer distribution: ";
	// for (int i = 0; i < nproc; ++i) {   
	//     total_counter += owner_counter[i];    
	//     cout << owner_counter[i] << " "; 
	//     // printf("GPU Supermer pack: output buffer: %d %d \n", owner_counter[i], total_counter);
	// }
	// cout << endl;

	hipFree(d_seq);
	hipFree(d_supermers);
	hipFree(d_slen);
	hipFree(d_owner_counter);

	cuda_timer_stop(start, stop, milliseconds);
	return;
}


__global__ void cu_kcounter_smer(KeyValue* hashtable, const keyType* kvs, const unsigned char* slens,  unsigned int numkvs, int klen, keyType mask)
{
	unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;

	if (threadid < numkvs){

		keyType new_smer = kvs[threadid];
		unsigned char c = slens[threadid];
		int slen = (int)c;

		for(int k = 0; k < (slen - klen + 1); ++k){
			
            keyType new_key = ((new_smer) >> (2*(31-(klen+k -1)))) & mask;//kvs[threadid];//.key;
			keyType slot = cuda_murmur3_64(new_key) & (kHashTableCapacity-1);

			while (true){
				keyType old_key = atomicCAS(&hashtable[slot].key, kEmpty, new_key);

				if (old_key == kEmpty || old_key == new_key) {
					atomicAdd(&hashtable[slot].value,1);
					break;
				}
				slot = (slot + 1) & (kHashTableCapacity-1);
			}
		}
	}
}

void kcounter_supermer_GPU(KeyValue* pHashTable, keyType* d_smers, unsigned char* d_slen, uint32_t num_keys, int klen, int rank)
{
	// Map MPI ranks to GPUs
	int count, devId;
	hipGetDeviceCount(&count);
	int gpuID = rank % count;
	hipSetDevice(gpuID);
	// hipGetDevice(&devId);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	int b = 128;
	// Have CUDA calculate the thread block size
	keyType mask = pow(2, 2 * klen) - 1;
	int mingridsize;
	int threadblocksize;
	hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, cu_kcounter_smer, 0, 0);

	unsigned char * h_slens = (unsigned char *) malloc(num_keys * sizeof(unsigned char));
	checkCuda (hipMemcpy(h_slens, d_slen, num_keys * sizeof(unsigned char), hipMemcpyDeviceToHost), __LINE__); 
	
	int gridsize = ((uint32_t)num_keys + threadblocksize - 1) / threadblocksize;
	cu_kcounter_smer<<<gridsize, threadblocksize>>>(pHashTable, d_smers, d_slen, (uint32_t)num_keys, klen, mask);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipFree(d_smers);
	hipFree(d_slen);
	return ;//h_pHashTable;

}



